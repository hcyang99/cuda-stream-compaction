#include "hip/hip_runtime.h"
#include "prefixScanGpu.cuh"
#include <stdio.h>

#define LOG_NUM_BANKS 5 
// #define CONFLICT_FREE_OFFSET(n) 0
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS) 


__global__ void 
prescan_single_block(uint32_t *g_odata, uint32_t *g_idata, uint32_t n) 
{ 
    int bx = blockIdx.x;
    if (bx != 0)
        return;
    
    n = 1024;

    __shared__ uint32_t temp[1024 + CONFLICT_FREE_OFFSET(1024)];  // allocated on invocation 

    int tx = threadIdx.x; 
    int offset = 1;

    int ai = 2 * tx;
    int bi = 2 * tx + 1;
    ai += CONFLICT_FREE_OFFSET(ai); 
    bi += CONFLICT_FREE_OFFSET(bi);
    temp[ai] = g_idata[2*tx]; 
    temp[bi] = g_idata[2*tx + 1]; 

    for (uint32_t d = n>>1; d > 0; d >>= 1)  // build sum in place up the tree 
    { 
        __syncthreads();
        int ai = offset*(2*tx+1)-1; 
        int bi = offset*(2*tx+2)-1;     
        if (bi < n) 
        {
            ai += CONFLICT_FREE_OFFSET(ai); 
            bi += CONFLICT_FREE_OFFSET(bi);  
            temp[bi] += temp[ai];    
        }    
        offset *= 2; 
    } 

    if (tx == 0) { temp[n - 1 + CONFLICT_FREE_OFFSET(n-1)] = 0; } // clear the last element
    for (uint32_t d = 1; d < n; d *= 2) // traverse down tree & build scan 
    {      
        offset >>= 1;      
        __syncthreads();
        int ai = offset*(2*tx+1)-1; 
        int bi = offset*(2*tx+2)-1;      
        if (bi < n) 
        {
            ai += CONFLICT_FREE_OFFSET(ai); 
            bi += CONFLICT_FREE_OFFSET(bi);   
            uint32_t t = temp[ai]; 
            temp[ai] = temp[bi]; 
            temp[bi] += t;       
        } 
    }  
    __syncthreads(); 

    // write results to device memory
    g_odata[2*tx] = temp[ai];
    g_odata[2*tx + 1] = temp[bi];
}

void prefixScanGpuSingleBlock(uint32_t* h_in, uint32_t* h_out, size_t length)
{
    if (length > 1024)
    {
        fprintf(stderr, "Invalid Parameter: length out of bound\n");
        exit(1);
    }

    uint32_t* d_in;
    uint32_t* d_out;

    auto err = hipMalloc(&d_in, 1024*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_out, 1024*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_in, h_in, 1024*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy 1 failed!\n");
        exit(1);
    }

    int numBlocks = 16; // i.e. number of thread blocks on the GPU
    int blockSize = 512; // i.e. number of GPU threads per thread block

    prescan_single_block<<<numBlocks, blockSize>>>(d_out, d_in, 1024);

    err = hipMemcpy(h_out, d_out, 1024*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy 2 failed!\n");
        fprintf(stderr, "%s\n", hipGetErrorName(err));
        fprintf(stderr, "h_out: %p, d_out: %p\n", h_out, d_out);
        exit(1);
    }

    hipFree(d_in);
    hipFree(d_out);
}