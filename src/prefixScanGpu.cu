#include "hip/hip_runtime.h"
#include "prefixScanGpu.cuh"
#include <stdio.h>

#define LOG_NUM_BANKS 5 
// #define CONFLICT_FREE_OFFSET(n) 0
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS) 


__global__ void 
prescan_single_block(uint32_t *g_odata, uint32_t *g_idata, uint32_t n) 
{ 
    int bx = blockIdx.x;
    if (bx != 0)
        return;
    
    n = 1024;

    __shared__ uint32_t temp[1024 + CONFLICT_FREE_OFFSET(1024)];  // allocated on invocation 

    int tx = threadIdx.x; 
    int offset = 1;

    int ai = 2 * tx;
    int bi = 2 * tx + 1;
    ai += CONFLICT_FREE_OFFSET(ai); 
    bi += CONFLICT_FREE_OFFSET(bi);
    temp[ai] = g_idata[2*tx]; 
    temp[bi] = g_idata[2*tx + 1]; 

    for (uint32_t d = n>>1; d > 0; d >>= 1)  // build sum in place up the tree 
    { 
        __syncthreads();
        int ai = offset*(2*tx+1)-1; 
        int bi = offset*(2*tx+2)-1;     
        if (bi < n) 
        {
            ai += CONFLICT_FREE_OFFSET(ai); 
            bi += CONFLICT_FREE_OFFSET(bi);  
            temp[bi] += temp[ai];    
        }    
        offset *= 2; 
    } 

    if (tx == 0) { temp[n - 1 + CONFLICT_FREE_OFFSET(n-1)] = 0; } // clear the last element
    for (uint32_t d = 1; d < n; d *= 2) // traverse down tree & build scan 
    {      
        offset >>= 1;      
        __syncthreads();
        int ai = offset*(2*tx+1)-1; 
        int bi = offset*(2*tx+2)-1;      
        if (bi < n) 
        {
            ai += CONFLICT_FREE_OFFSET(ai); 
            bi += CONFLICT_FREE_OFFSET(bi);   
            uint32_t t = temp[ai]; 
            temp[ai] = temp[bi]; 
            temp[bi] += t;       
        } 
    }  
    __syncthreads(); 

    // write results to device memory
    g_odata[2*tx] = temp[ai];
    g_odata[2*tx + 1] = temp[bi];
}

__global__ void 
prescan_partial(uint32_t *g_odata, uint32_t *g_idata, uint32_t n) 
{ 
    __shared__ uint32_t temp[1024 + CONFLICT_FREE_OFFSET(1024)];  // allocated on invocation 

    int bx = blockIdx.x;
    int tx = threadIdx.x; 
    int gs = gridDim.x;
    
    int batch_size = n / 1024 / gs + 1;
    int batch_start = bx * batch_size;
    int batch_end = (bx + 1) * batch_size;
    if (batch_end > n / 1024) batch_end = n / 1024;

    for (int batch = batch_start; batch < batch_end; ++batch)
    {
        int offset = 1;

        int ai = 2 * tx;
        int bi = 2 * tx + 1;
        ai += CONFLICT_FREE_OFFSET(ai); 
        bi += CONFLICT_FREE_OFFSET(bi);
        temp[ai] = g_idata[2*tx + batch * 1024]; 
        temp[bi] = g_idata[2*tx + 1 + batch * 1024]; 

        for (uint32_t d = 1024>>1; d > 0; d >>= 1)  // build sum in place up the tree 
        { 
            __syncthreads();
            int ai = offset*(2*tx+1)-1; 
            int bi = offset*(2*tx+2)-1;     
            if (bi < 1024) 
            {
                ai += CONFLICT_FREE_OFFSET(ai); 
                bi += CONFLICT_FREE_OFFSET(bi);  
                temp[bi] += temp[ai];    
            }    
            offset *= 2; 
        } 

        if (tx == 0) { temp[1024 - 1 + CONFLICT_FREE_OFFSET(1024-1)] = 0; } // clear the last element
        for (uint32_t d = 1; d < 1024; d *= 2) // traverse down tree & build scan 
        {      
            offset >>= 1;      
            __syncthreads();
            int ai = offset*(2*tx+1)-1; 
            int bi = offset*(2*tx+2)-1;      
            if (bi < 1024) 
            {
                ai += CONFLICT_FREE_OFFSET(ai); 
                bi += CONFLICT_FREE_OFFSET(bi);   
                uint32_t t = temp[ai]; 
                temp[ai] = temp[bi]; 
                temp[bi] += t;       
            } 
        }  
        __syncthreads(); 

        // write results to device memory
        g_odata[2*tx + batch * 1024] = temp[ai];
        g_odata[2*tx + 1 + batch * 1024] = temp[bi];
    }
}

void prefixScanGpuSingleBlock(uint32_t* h_in, uint32_t* h_out, size_t length)
{
    if (length > 1024)
    {
        fprintf(stderr, "Invalid Parameter: length out of bound\n");
        exit(1);
    }

    uint32_t* d_in;
    uint32_t* d_out;

    auto err = hipMalloc(&d_in, length*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_out, length*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_in, h_in, length*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy 1 failed!\n");
        exit(1);
    }

    int numBlocks = 128; // i.e. number of thread blocks on the GPU
    int blockSize = 512; // i.e. number of GPU threads per thread block

    prescan_single_block<<<numBlocks, blockSize>>>(d_out, d_in, 1024);

    err = hipMemcpy(h_out, d_out, length*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy 2 failed!\n");
        fprintf(stderr, "%s\n", hipGetErrorName(err));
        fprintf(stderr, "h_out: %p, d_out: %p\n", h_out, d_out);
        exit(1);
    }

    hipFree(d_in);
    hipFree(d_out);
}

void prefixScanGpuPartial(uint32_t* h_in, uint32_t* h_out, size_t length)
{
    if (length % 1024 != 0)
    {
        fprintf(stderr, "Invalid Parameter: length not aligned to 1024\n");
        exit(1);
    }

    uint32_t* d_in;
    uint32_t* d_out;

    auto err = hipMalloc(&d_in, length*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_out, length*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_in, h_in, length*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy 1 failed!\n");
        exit(1);
    }

    int numBlocks = 128; // i.e. number of thread blocks on the GPU
    int blockSize = 512; // i.e. number of GPU threads per thread block

    prescan_partial<<<numBlocks, blockSize>>>(d_out, d_in, length);

    err = hipMemcpy(h_out, d_out, length*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy 2 failed!\n");
        fprintf(stderr, "%s\n", hipGetErrorName(err));
        fprintf(stderr, "h_out: %p, d_out: %p\n", h_out, d_out);
        exit(1);
    }

    hipFree(d_in);
    hipFree(d_out);
}