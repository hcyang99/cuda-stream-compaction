#include "hip/hip_runtime.h"
#include "prefixScanGpu.cuh"
#include <stdio.h>

#define LOG_NUM_BANKS 5 
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS) 

/**
 * Prints information for each available GPU device on stdout
 */
void printGpuProperties () {
    int nDevices;

    // Store the number of available GPU device in nDevicess
    hipError_t err = hipGetDeviceCount(&nDevices);

    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipGetDeviceCount failed!\n");
        exit(1);
    }

    // For each GPU device found, print the information (memory, bandwidth etc.)
    // about the device
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Device memory: %lu\n", prop.totalGlobalMem);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
}


__global__ void 
prescan_single_block(uint32_t *g_odata, uint32_t *g_idata, uint32_t n) 
{ 
    int bx = blockIdx.x;
    if (bx != 0)
        return;
    
    n = 1024;

    __shared__ uint32_t temp[1024 + CONFLICT_FREE_OFFSET(1024)];  // allocated on invocation 

    int tx = threadIdx.x; 
    int offset = 1;

    int ai = 2 * tx;
    int bi = 2 * tx + 1;
    ai += CONFLICT_FREE_OFFSET(ai); 
    bi += CONFLICT_FREE_OFFSET(bi);
    temp[ai] = g_idata[2*tx]; 
    temp[bi] = g_idata[2*tx + 1]; 

    for (uint32_t d = n>>1; d > 0; d >>= 1)  // build sum in place up the tree 
    { 
        __syncthreads();
        int ai = offset*(2*tx+1)-1; 
        int bi = offset*(2*tx+2)-1;     
        if (bi < n) 
        {
            ai += CONFLICT_FREE_OFFSET(ai); 
            bi += CONFLICT_FREE_OFFSET(bi);  
            temp[bi] += temp[ai];    
        }    
        offset *= 2; 
    } 

    if (tx == 0) { temp[n - 1 + CONFLICT_FREE_OFFSET(n-1)] = 0; } // clear the last element
    for (uint32_t d = 1; d < n; d *= 2) // traverse down tree & build scan 
    {      
        offset >>= 1;      
        __syncthreads();
        int ai = offset*(2*tx+1)-1; 
        int bi = offset*(2*tx+2)-1;      
        if (bi < n) 
        {
            ai += CONFLICT_FREE_OFFSET(ai); 
            bi += CONFLICT_FREE_OFFSET(bi);   
            uint32_t t = temp[ai]; 
            temp[ai] = temp[bi]; 
            temp[bi] += t;       
        } 
    }  
    __syncthreads(); 

    // write results to device memory
    g_odata[2*tx] = temp[ai];
    g_odata[2*tx + 1] = temp[bi];
}

__global__ void 
prescan_partial(uint32_t *g_odata, uint32_t *g_idata, uint32_t n) 
{ 
    __shared__ uint32_t temp[1024 + CONFLICT_FREE_OFFSET(1024)];  // allocated on invocation 

    int bx = blockIdx.x;
    int tx = threadIdx.x; 
    int gs = gridDim.x;
    
    int batch_size = n / 1024 / gs + 1;
    int batch_start = bx * batch_size;
    int batch_end = (bx + 1) * batch_size;
    if (batch_end > n / 1024) batch_end = n / 1024;

    for (int batch = batch_start; batch < batch_end; ++batch)
    {
        int offset = 1;

        int ai = 2 * tx;
        int bi = 2 * tx + 1;
        ai += CONFLICT_FREE_OFFSET(ai); 
        bi += CONFLICT_FREE_OFFSET(bi);
        temp[ai] = g_idata[2*tx + batch * 1024]; 
        temp[bi] = g_idata[2*tx + 1 + batch * 1024]; 

        for (uint32_t d = 1024>>1; d > 0; d >>= 1)  // build sum in place up the tree 
        { 
            __syncthreads();
            int ai = offset*(2*tx+1)-1; 
            int bi = offset*(2*tx+2)-1;     
            if (bi < 1024) 
            {
                ai += CONFLICT_FREE_OFFSET(ai); 
                bi += CONFLICT_FREE_OFFSET(bi);  
                temp[bi] += temp[ai];    
            }    
            offset *= 2; 
        } 

        if (tx == 0) { temp[1024 - 1 + CONFLICT_FREE_OFFSET(1024-1)] = 0; } // clear the last element
        for (uint32_t d = 1; d < 1024; d *= 2) // traverse down tree & build scan 
        {      
            offset >>= 1;      
            __syncthreads();
            int ai = offset*(2*tx+1)-1; 
            int bi = offset*(2*tx+2)-1;      
            if (bi < 1024) 
            {
                ai += CONFLICT_FREE_OFFSET(ai); 
                bi += CONFLICT_FREE_OFFSET(bi);   
                uint32_t t = temp[ai]; 
                temp[ai] = temp[bi]; 
                temp[bi] += t;       
            } 
        }  
        __syncthreads(); 

        // write results to device memory
        g_odata[2*tx + batch * 1024] = temp[ai];
        g_odata[2*tx + 1 + batch * 1024] = temp[bi];
    }
}

__global__ void
prescan_add(uint32_t* d_out, uint32_t* d_in, uint32_t num_batches)
{
    int bx = blockIdx.x;
    int tx = threadIdx.x; 
    int gs = gridDim.x;
    int bs = blockDim.x;

    int batch_size = num_batches / gs + 1;
    int batch_start = bx * batch_size;
    int batch_end = (bx + 1) * batch_size;
    if (batch_end > num_batches) batch_end = num_batches;

    for (uint32_t i = batch_start * 1024 + tx; i < batch_end * 1024; i += bs)
    {
        d_out[i] += d_in[i / 1024];
    }
}

__global__ void
prescan_sum(uint32_t* d_out, uint32_t* d_in, uint32_t num_batches)
{
    __shared__ uint32_t s[32];
    int bx = blockIdx.x;
    int tx = threadIdx.x; 
    int gs = gridDim.x;
    int bs = blockDim.x;

    int batch_size = num_batches / gs + 1;
    int batch_start = bx * batch_size;
    int batch_end = (bx + 1) * batch_size;
    if (batch_end > num_batches) batch_end = num_batches;

    for (int batch = batch_start; batch < batch_end; ++batch)
    {
        uint32_t local_sum = 0;
        for (int i = tx; i < 1024; i += bs)
        {
            local_sum += d_in[i + batch * 1024];
        }
        s[tx] = local_sum;
        __syncthreads();
        if (tx == 0)
        {
            uint32_t master_sum = 0;
            for (int i = 0; i < bs; ++i)
            {
                master_sum += s[i];
            }
            d_out[batch] = master_sum;
        }
        __syncthreads();
    }

    if (bx == gs - 1)
    {
        uint32_t num_batch_aligned = num_batches;
        if (num_batches % 1024 != 0) num_batch_aligned = (num_batches / 1024 + 1) * 1024;
        for (int i = num_batches + tx; i < num_batch_aligned; i += 32)
            d_out[i] = 0;
    }
}

__global__ void
gen_mask(uint64_t* d_in, uint32_t* d_mask_out, size_t length)
{
    int bx = blockIdx.x;
    int tx = threadIdx.x; 
    int gs = gridDim.x;
    int bs = blockDim.x;

    size_t aligned = (length % 1024) ? (length / 1024 + 1) * 1024 : length;

    size_t partition_size = length / gs + 1;
    if (partition_size % bs != 0) partition_size = (partition_size / bs + 1) * bs;
    size_t begin = partition_size * bx;
    size_t end = partition_size * (bx + 1);
    if (begin == 0) begin = 1;
    if (end > length) end = length;

    if (bx == 0 && tx == 0)
        d_mask_out[0] = 1;
    for (int i = begin + tx; i < end; i += bs)
    {
        d_mask_out[i] = (d_in[i-1] == d_in[i]) ? 0 : 1;
    }
    if (bx == gs - 1)
    {
        for (int i = length + tx; i < aligned; i += bs)
            d_mask_out[i] = 0;
    }
}

__global__ void
masked_scatter(uint64_t* d_in, uint64_t* d_out, uint32_t* d_mask, uint32_t* d_addr, size_t length)
{
    int bx = blockIdx.x;
    int tx = threadIdx.x; 
    int gs = gridDim.x;
    int bs = blockDim.x;

    size_t partition_size = length / gs + 1;
    size_t begin = partition_size * bx;
    size_t end = partition_size * (bx + 1);
    if (end > length) end = length;

    if (bx == 0 && tx == 0)
        if (d_mask[0])
        {
            d_out[d_addr[0]] = d_in[0];
        }

    for (int i = begin + tx; i < end; i += bs)
    {
        if (d_mask[i])
        {
            d_out[d_addr[i]] = d_in[i];
        }
    }
}

void prefixScanGpu(uint32_t* d_in, uint32_t* d_out, size_t length)
{
    if (length <= 1024)
    {
        // fprintf(stderr, "Start Prescan with 1 block\n");
        prescan_single_block<<<1, 512>>>(d_out, d_in, 1024);
    } 
    else 
    {
        // call partial scan
        size_t aligned_length;
        if (length % 1024 == 0) aligned_length = length; else aligned_length = (length / 1024 + 1) * 1024;
        prescan_partial<<<128,512>>>(d_out, d_in, aligned_length);

        // call sum
        size_t sum_length, sum_aligned;
        uint32_t* sum_out;
        sum_length = aligned_length / 1024;
        if (sum_length % 1024 == 0) sum_aligned = sum_length; else sum_aligned = (sum_length / 1024 + 1) * 1024;
        auto err = hipMalloc(&sum_out, sum_aligned*sizeof(uint32_t));
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        prescan_sum<<<512,32>>>(sum_out, d_in, sum_length);

        // call recursive scan
        uint32_t* sum_scanned;
        err = hipMalloc(&sum_scanned, sum_aligned*sizeof(uint32_t));
        if (err != hipSuccess) {
            fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
            exit(1);
        }
        prefixScanGpu(sum_out, sum_scanned, sum_length);
 

        // call scatter add
        prescan_add<<<128, 512>>>(d_out, sum_scanned, sum_length);

        // free memory
        hipFree(sum_out);
        hipFree(sum_scanned);
    }
}

void stream_compaction(uint64_t** d_out_ptr, size_t* out_length_ptr, uint64_t* d_in, size_t length)
{   
    size_t aligned = length;
    if (aligned % 1024 != 0) aligned = (aligned / 1024 + 1) * 1024;

    // generate the mask
    uint32_t* d_mask;
    auto err = hipMalloc(&d_mask, aligned*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }
    gen_mask<<<512, 128>>>(d_in, d_mask, length);

    // generate the address
    uint32_t* d_addr;
    err = hipMalloc(&d_addr, aligned*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }
    prefixScanGpu(d_mask, d_addr, length);

    // allocate the output
    uint64_t* d_out;
    uint32_t out_length;
    uint32_t last_mask;
    err = hipMemcpy(&out_length, d_addr + length - 1, sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    err = hipMemcpy(&last_mask, d_mask + length - 1, sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }
    out_length += last_mask;

    err = hipMalloc(&d_out, out_length*sizeof(uint64_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    *d_out_ptr = d_out;
    *out_length_ptr = out_length;


    // write to the output
    masked_scatter<<<512,128>>>(d_in, d_out, d_mask, d_addr, length);

    // deallocate memory
    hipFree(d_mask);
    hipFree(d_addr);
}

void testMaskedScatterGpu(uint64_t* h_in, uint64_t* h_out, uint32_t* h_mask, uint32_t* h_addr, size_t length)
{
    uint64_t* d_in;
    uint64_t* d_out;
    uint32_t* d_mask;
    uint32_t* d_addr;
    size_t out_length = h_addr[length - 1] + h_mask[length - 1];

    auto err = hipMalloc(&d_in, length*sizeof(uint64_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_out, out_length*sizeof(uint64_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_mask, length*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_addr, length*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_in, h_in, length*sizeof(uint64_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_mask, h_mask, length*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_addr, h_addr, length*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    masked_scatter<<<512,128>>>(d_in, d_out, d_mask, d_addr, length);

    err = hipMemcpy(h_out, d_out, out_length*sizeof(uint64_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }
}

void testMaskGenGpu(uint64_t* h_in, uint32_t* h_out, size_t size)
{
    uint64_t* d_in;
    uint32_t* d_out;

    auto err = hipMalloc(&d_in, size*sizeof(uint64_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_out, size*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_in, h_in, size*sizeof(uint64_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    gen_mask<<<512, 128>>>(d_in, d_out, size);

    err = hipMemcpy(h_out, d_out, size*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }
}

void testStreamCompactionGpu(uint64_t* h_in, uint64_t* h_out, size_t size)
{
    uint64_t* d_in;
    uint64_t* d_out;
    size_t out_length;

    auto err = hipMalloc(&d_in, size*sizeof(uint64_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }
    err = hipMemcpy(d_in, h_in, size*sizeof(uint64_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        fprintf(stderr, "%s\n", hipGetErrorName(err));
        exit(1);
    }

    stream_compaction(&d_out, &out_length, d_in, size);

    err = hipMemcpy(h_out, d_out, out_length*sizeof(uint64_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    hipFree(d_in);
    hipFree(d_out);
}

void testPrefixScanGpu(uint32_t* h_in, uint32_t* h_out, size_t size)
{
    uint32_t* d_in;
    uint32_t* d_out;
    uint32_t aligned;
    if (size % 1024 != 0)
        aligned = (size / 1024 + 1) * 1024;
    else 
        aligned = size;

    auto err = hipMalloc(&d_in, aligned*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }
    err = hipMalloc(&d_out, aligned*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }
    err = hipMemcpy(d_in, h_in, aligned*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    prefixScanGpu(d_in, d_out, size);

    err = hipMemcpy(h_out, d_out, aligned*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    hipFree(d_in);
    hipFree(d_out);
}


void prefixScanGpuSingleBlock(uint32_t* h_in, uint32_t* h_out, size_t length)
{
    if (length > 1024)
    {
        fprintf(stderr, "Invalid Parameter: length out of bound\n");
        exit(1);
    }

    uint32_t* d_in;
    uint32_t* d_out;

    auto err = hipMalloc(&d_in, length*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_out, length*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_in, h_in, length*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy 1 failed!\n");
        exit(1);
    }

    int numBlocks = 128; // i.e. number of thread blocks on the GPU
    int blockSize = 512; // i.e. number of GPU threads per thread block

    prescan_single_block<<<numBlocks, blockSize>>>(d_out, d_in, 1024);

    err = hipMemcpy(h_out, d_out, length*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy 2 failed!\n");
        fprintf(stderr, "%s\n", hipGetErrorName(err));
        fprintf(stderr, "h_out: %p, d_out: %p\n", h_out, d_out);
        exit(1);
    }

    hipFree(d_in);
    hipFree(d_out);
}

void prefixScanGpuPartial(uint32_t* h_in, uint32_t* h_out, size_t length)
{
    if (length % 1024 != 0)
    {
        fprintf(stderr, "Invalid Parameter: length not aligned to 1024\n");
        exit(1);
    }

    uint32_t* d_in;
    uint32_t* d_out;

    auto err = hipMalloc(&d_in, length*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_out, length*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_in, h_in, length*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy 1 failed!\n");
        exit(1);
    }

    int numBlocks = 128; // i.e. number of thread blocks on the GPU
    int blockSize = 512; // i.e. number of GPU threads per thread block

    prescan_partial<<<numBlocks, blockSize>>>(d_out, d_in, length);

    err = hipMemcpy(h_out, d_out, length*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy 2 failed!\n");
        fprintf(stderr, "%s\n", hipGetErrorName(err));
        fprintf(stderr, "h_out: %p, d_out: %p\n", h_out, d_out);
        exit(1);
    }

    hipFree(d_in);
    hipFree(d_out);
}